#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#define BOX_SIZE 1024

#include "hip/hip_runtime.h"
#include ""
#include "simple_knn.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#define __HIPCC__
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

struct CustomMin
{
	__device__ __forceinline__
		float3 operator()(const float3& a, const float3& b) const {
		return { min(a.x, b.x), min(a.y, b.y), min(a.z, b.z) };
	}
};

struct CustomMax
{
	__device__ __forceinline__
		float3 operator()(const float3& a, const float3& b) const {
		return { max(a.x, b.x), max(a.y, b.y), max(a.z, b.z) };
	}
};

__host__ __device__ uint32_t prepMorton(uint32_t x)
{
	x = (x | (x << 16)) & 0x030000FF;
	x = (x | (x << 8)) & 0x0300F00F;
	x = (x | (x << 4)) & 0x030C30C3;
	x = (x | (x << 2)) & 0x09249249;
	return x;
}

__host__ __device__ uint32_t coord2Morton(float3 coord, float3 minn, float3 maxx)
{
	uint32_t x = prepMorton(((coord.x - minn.x) / (maxx.x - minn.x)) * ((1 << 10) - 1));
	uint32_t y = prepMorton(((coord.y - minn.y) / (maxx.y - minn.y)) * ((1 << 10) - 1));
	uint32_t z = prepMorton(((coord.z - minn.z) / (maxx.z - minn.z)) * ((1 << 10) - 1));

	return x | (y << 1) | (z << 2);
}

__global__ void coord2Morton(int P, const float3* points, float3 minn, float3 maxx, uint32_t* codes)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	codes[idx] = coord2Morton(points[idx], minn, maxx);
}

struct MinMax
{
	float3 minn;
	float3 maxx;
};

__global__ void boxMinMax(uint32_t P, float3* points, uint32_t* indices, MinMax* boxes)
{
	auto idx = cg::this_grid().thread_rank();

	MinMax me;
	if (idx < P)
	{
		me.minn = points[indices[idx]];
		me.maxx = points[indices[idx]];
	}
	else
	{
		me.minn = { FLT_MAX, FLT_MAX, FLT_MAX };
		me.maxx = { -FLT_MAX,-FLT_MAX,-FLT_MAX };
	}

	__shared__ MinMax redResult[BOX_SIZE];

	for (int off = BOX_SIZE / 2; off >= 1; off /= 2)
	{
		if (threadIdx.x < 2 * off)
			redResult[threadIdx.x] = me;
		__syncthreads();

		if (threadIdx.x < off)
		{
			MinMax other = redResult[threadIdx.x + off];
			me.minn.x = min(me.minn.x, other.minn.x);
			me.minn.y = min(me.minn.y, other.minn.y);
			me.minn.z = min(me.minn.z, other.minn.z);
			me.maxx.x = max(me.maxx.x, other.maxx.x);
			me.maxx.y = max(me.maxx.y, other.maxx.y);
			me.maxx.z = max(me.maxx.z, other.maxx.z);
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
		boxes[blockIdx.x] = me;
}

__device__ __host__ float distBoxPoint(const MinMax& box, const float3& p)
{
	float3 diff = { 0, 0, 0 };
	if (p.x < box.minn.x || p.x > box.maxx.x)
		diff.x = min(abs(p.x - box.minn.x), abs(p.x - box.maxx.x));
	if (p.y < box.minn.y || p.y > box.maxx.y)
		diff.y = min(abs(p.y - box.minn.y), abs(p.y - box.maxx.y));
	if (p.z < box.minn.z || p.z > box.maxx.z)
		diff.z = min(abs(p.z - box.minn.z), abs(p.z - box.maxx.z));
	return diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
}

template<int K>
__device__ void updateKBest(const float3& ref, const float3& point, float* knn)
{
	float3 d = { point.x - ref.x, point.y - ref.y, point.z - ref.z };
	float dist = d.x * d.x + d.y * d.y + d.z * d.z;
	for (int j = 0; j < K; j++)
	{
		if (knn[j] > dist)
		{
			float t = knn[j];
			knn[j] = dist;
			dist = t;
		}
	}
}

__global__ void boxMeanDist(uint32_t P, float3* points, uint32_t* indices, MinMax* boxes, float* dists)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 point = points[indices[idx]];
	float best[3] = { FLT_MAX, FLT_MAX, FLT_MAX };

	for (int i = max(0, idx - 3); i <= min(P - 1, idx + 3); i++)
	{
		if (i == idx)
			continue;
		updateKBest<3>(point, points[indices[i]], best);
	}

	float reject = best[2];
	best[0] = FLT_MAX;
	best[1] = FLT_MAX;
	best[2] = FLT_MAX;

	for (int b = 0; b < (P + BOX_SIZE - 1) / BOX_SIZE; b++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist > reject || dist > best[2])
			continue;

		for (int i = b * BOX_SIZE; i < min(P, (b + 1) * BOX_SIZE); i++)
		{
			if (i == idx)
				continue;
			updateKBest<3>(point, points[indices[i]], best);
		}
	}
	dists[indices[idx]] = (best[0] + best[1] + best[2]) / 3.0f;
}

void SimpleKNN::knn(int P, float3* points, float* meanDists)
{
	float3* result;
	hipMalloc(&result, sizeof(float3));
	size_t temp_storage_bytes;

	float3 init = { 0, 0, 0 }, minn, maxx;

	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);

	thrust::device_vector<uint32_t> morton(P);
	thrust::device_vector<uint32_t> morton_sorted(P);
	coord2Morton << <(P + 255) / 256, 256 >> > (P, points, minn, maxx, morton.data().get());

	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);

	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	uint32_t num_boxes = (P + BOX_SIZE - 1) / BOX_SIZE;
	thrust::device_vector<MinMax> boxes(num_boxes);
	boxMinMax << <num_boxes, BOX_SIZE >> > (P, points, indices_sorted.data().get(), boxes.data().get());
	boxMeanDist << <num_boxes, BOX_SIZE >> > (P, points, indices_sorted.data().get(), boxes.data().get(), meanDists);

	hipFree(result);
}